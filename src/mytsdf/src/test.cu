#include "hip/hip_runtime.h"
#include "common/book.h"

__global__ void add(int a,int b,int *c)
{
    *c = a + b;
}

int tsdf()
{
//    hipDeviceProp_t  prop;
//    int dev;

//    HANDLE_ERROR( hipGetDevice( &dev ) );
//    printf( "ID of current CUDA device:  %d\n", dev );

//    memset( &prop, 0, sizeof( hipDeviceProp_t ) );
//    prop.major = 1;
//    prop.minor = 3;
//    HANDLE_ERROR( hipChooseDevice( &dev, &prop ) );
//    printf( "ID of CUDA device closest to revision 1.3:  %d\n", dev );

//    HANDLE_ERROR( hipSetDevice( dev ) );

    int c;
    int *dev_c;
    HANDLE_ERROR( hipMalloc( (void**)&dev_c, sizeof(int) ) );
    add<<<1,1>>>(2,7,dev_c);
    HANDLE_ERROR( hipMemcpy( &c, dev_c, sizeof(int),
                              hipMemcpyDeviceToHost ) );
    printf( "2 + 7 = %d\n", c );
    HANDLE_ERROR( hipFree( dev_c ) );

    return 0;
}
