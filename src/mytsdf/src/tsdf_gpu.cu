#include "hip/hip_runtime.h"
#include "tsdf_gpu.cuh"

// CUDA kernel function to integrate a TSDF voxel volume given depth images
__global__
void Integrate(float * cam_K, float * cam2world, float * depth_im, unsigned char * rgb_im,
               int im_height, int im_width, int voxel_grid_dim_x, int voxel_grid_dim_y, int voxel_grid_dim_z,
               float voxel_grid_origin_x, float voxel_grid_origin_y, float voxel_grid_origin_z, float voxel_size, float trunc_margin,
               float * voxel_grid_TSDF, float * voxel_grid_weight,
               unsigned char * voxel_grid_rgb, float * voxel_grid_rgb_weight, float * voxel_grid_rgb_diff) {

  int pt_grid_z = blockIdx.x;
  int pt_grid_y = threadIdx.x;

  for (int pt_grid_x = 0; pt_grid_x < voxel_grid_dim_x; ++pt_grid_x) {

    // Convert voxel center from grid coordinates to base frame camera coordinates
    float pt_base_x = voxel_grid_origin_x + pt_grid_x * voxel_size;
    float pt_base_y = voxel_grid_origin_y + pt_grid_y * voxel_size;
    float pt_base_z = voxel_grid_origin_z + pt_grid_z * voxel_size;

    // Convert from base frame camera coordinates to current frame camera coordinates
    float tmp_pt[3] = {0};
    tmp_pt[0] = pt_base_x - cam2world[0 * 4 + 3];
    tmp_pt[1] = pt_base_y - cam2world[1 * 4 + 3];
    tmp_pt[2] = pt_base_z - cam2world[2 * 4 + 3];
    float pt_cam_x = cam2world[0 * 4 + 0] * tmp_pt[0] + cam2world[1 * 4 + 0] * tmp_pt[1] + cam2world[2 * 4 + 0] * tmp_pt[2];
    float pt_cam_y = cam2world[0 * 4 + 1] * tmp_pt[0] + cam2world[1 * 4 + 1] * tmp_pt[1] + cam2world[2 * 4 + 1] * tmp_pt[2];
    float pt_cam_z = cam2world[0 * 4 + 2] * tmp_pt[0] + cam2world[1 * 4 + 2] * tmp_pt[1] + cam2world[2 * 4 + 2] * tmp_pt[2];

    if (pt_cam_z <= 0)
      continue;

    int pt_pix_x = roundf(cam_K[0 * 3 + 0] * (pt_cam_x / pt_cam_z) + cam_K[0 * 3 + 2]);
    int pt_pix_y = roundf(cam_K[1 * 3 + 1] * (pt_cam_y / pt_cam_z) + cam_K[1 * 3 + 2]);
    if (pt_pix_x < 0 || pt_pix_x >= im_width || pt_pix_y < 0 || pt_pix_y >= im_height)
      continue;

    float depth_val = depth_im[pt_pix_y * im_width + pt_pix_x];
    unsigned char rgb_val[3] = {0};
    rgb_val[0] = rgb_im[pt_pix_y * im_width * 3 + pt_pix_x * 3 + 2];
    rgb_val[1] = rgb_im[pt_pix_y * im_width * 3 + pt_pix_x * 3 + 1];
    rgb_val[2] = rgb_im[pt_pix_y * im_width * 3 + pt_pix_x * 3];

    if (depth_val <= 0 || depth_val > 6)
      continue;

//    float diff = (depth_val - pt_cam_z) * sqrtf(1 + powf((pt_cam_x / pt_cam_z), 2) + powf((pt_cam_y / pt_cam_z), 2));
    float diff = depth_val - pt_cam_z;
    if (diff <= -trunc_margin)
      continue;

    // Integrate
    int volume_idx = pt_grid_z * voxel_grid_dim_y * voxel_grid_dim_x + pt_grid_y * voxel_grid_dim_x + pt_grid_x;
    float dist = fmin(1.0f, diff / trunc_margin);
    float weight_old = voxel_grid_weight[volume_idx];
    float weight_new = weight_old + 1.0f;
    voxel_grid_weight[volume_idx] = weight_new;
    voxel_grid_TSDF[volume_idx] = (voxel_grid_TSDF[volume_idx] * weight_old + dist) / weight_new;

    if((rgb_val[0] == 0)&&(rgb_val[1] == 0)&&(rgb_val[2] == 0))
      continue;

    if(voxel_grid_rgb_weight[volume_idx] == 0)
    {
      voxel_grid_rgb[volume_idx * 3] = rgb_val[0];
      voxel_grid_rgb[volume_idx * 3 + 1] = rgb_val[1];
      voxel_grid_rgb[volume_idx * 3 + 2] = rgb_val[2];
      voxel_grid_rgb_weight[volume_idx] = voxel_grid_rgb_weight[volume_idx] + 1.0f; 
      continue;
    }
    float rgb_weight_old = voxel_grid_rgb_weight[volume_idx];
    float rgb_weight_new = rgb_weight_old + 1.0f;
    voxel_grid_rgb_weight[volume_idx] = rgb_weight_new;
    float rgb_diff = std::abs(voxel_grid_rgb[volume_idx * 3] - rgb_val[0]) + 
                     std::abs(voxel_grid_rgb[volume_idx * 3 + 1] - rgb_val[1]) + 
                     std::abs(voxel_grid_rgb[volume_idx * 3 + 2] - rgb_val[2]);
    voxel_grid_rgb_diff[volume_idx] = (voxel_grid_rgb_diff[volume_idx] * rgb_weight_old + rgb_diff) / rgb_weight_new;
    // std::cout<<"rgb_weight:"<<rgb_weight_new<<" rgb_diff:"<<voxel_grid_rgb_diff[volume_idx]<<std::endl;
    if(voxel_grid_rgb_diff[volume_idx]>90)
    {
      voxel_grid_rgb[volume_idx * 3] = rgb_val[0];
      voxel_grid_rgb[volume_idx * 3 + 1] = rgb_val[1];
      voxel_grid_rgb[volume_idx * 3 + 2] = rgb_val[2];
    }

  }
}

// Compute surface points from TSDF voxel grid and save points to point cloud file
void SaveVoxelGrid2SurfacePointCloud(const std::string &file_name, int voxel_grid_dim_x, int voxel_grid_dim_y, int voxel_grid_dim_z,
                                     float voxel_size, float voxel_grid_origin_x, float voxel_grid_origin_y, float voxel_grid_origin_z,
                                     float * voxel_grid_TSDF, float * voxel_grid_weight, unsigned char * voxel_grid_rgb,
                                     float tsdf_thresh, float weight_thresh) {

  // Count total number of points in point cloud
  int num_pts = 0;
  for (int i = 0; i < voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z; i++)
    if (std::abs(voxel_grid_TSDF[i]) < tsdf_thresh && voxel_grid_weight[i] > weight_thresh)
      num_pts++;

  // Create header for .ply file
  FILE *fp = fopen(file_name.c_str(), "w");
  fprintf(fp, "ply\n");
  fprintf(fp, "format binary_little_endian 1.0\n");
  // fprintf(fp, "format ascii 1.0\n");
  fprintf(fp, "element vertex %d\n", num_pts);
  fprintf(fp, "property float x\n");
  fprintf(fp, "property float y\n");
  fprintf(fp, "property float z\n");
  fprintf(fp, "property uchar red\n");
  fprintf(fp, "property uchar green\n");
  fprintf(fp, "property uchar blue\n");
  fprintf(fp, "end_header\n");

  // Create point cloud content for ply file
  for (int i = 0; i < voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z; i++) {

    // If TSDF value of voxel is less than some threshold, add voxel coordinates to point cloud
    if (std::abs(voxel_grid_TSDF[i]) < tsdf_thresh && voxel_grid_weight[i] > weight_thresh) {

      // Compute voxel indices in int for higher positive number range
      int z = floor(i / (voxel_grid_dim_x * voxel_grid_dim_y));
      int y = floor((i - (z * voxel_grid_dim_x * voxel_grid_dim_y)) / voxel_grid_dim_x);
      int x = i - (z * voxel_grid_dim_x * voxel_grid_dim_y) - (y * voxel_grid_dim_x);

      // Convert voxel indices to float, and save coordinates to ply file
      float pt_base_x = voxel_grid_origin_x + (float) x * voxel_size;
      float pt_base_y = voxel_grid_origin_y + (float) y * voxel_size;
      float pt_base_z = voxel_grid_origin_z + (float) z * voxel_size;
      unsigned char pt_base_r = voxel_grid_rgb[i * 3];
      unsigned char pt_base_g = voxel_grid_rgb[i * 3 + 1];
      unsigned char pt_base_b = voxel_grid_rgb[i * 3 + 2];
      fwrite(&pt_base_x, sizeof(float), 1, fp);
      fwrite(&pt_base_y, sizeof(float), 1, fp);
      fwrite(&pt_base_z, sizeof(float), 1, fp);
      fwrite(&pt_base_r, sizeof(unsigned char), 1, fp);
      fwrite(&pt_base_g, sizeof(unsigned char), 1, fp);
      fwrite(&pt_base_b, sizeof(unsigned char), 1, fp);
      // if((pt_base_r<50)&&(pt_base_g<50)&&(pt_base_b<50))
      // {
      //   std::cout<<"r:"<<(int)pt_base_r;
      //   std::cout<<" g:"<<(int)pt_base_g;
      //   std::cout<<" b:"<<(int)pt_base_b<<std::endl;
      // }
      // std::cout<<"No: i"<<i<<" r:"<<(int)pt_base_r<<" g:"<< (int)pt_base_g<<" b:"<<(int)pt_base_r<<std::endl;
    }
  }
  fclose(fp);
}

// Load an M x N matrix from a text file (numbers delimited by spaces/tabs)
// Return the matrix as a float vector of the matrix in row-major order
std::vector<float> LoadMatrixFromFile(std::string filename, int M, int N) {
  std::vector<float> matrix;
  FILE *fp = fopen(filename.c_str(), "r");
  for (int i = 0; i < M * N; i++) {
    float tmp;
    int iret = fscanf(fp, "%f", &tmp);
    matrix.push_back(tmp);
  }
  fclose(fp);
  return matrix;
}

std::vector<float> Matrix2Array(cv::Matx33f input, int M, int N)
{
  std::vector<float> array;
  for (int i = 0; i < M; i++)
  {
    for(int j=0;j<N;j++)
    {
        float tmp;
        tmp = input(i,j);
        array.push_back(tmp);
    }
  }
  return array;
}

std::vector<float> Matrix2Array(Eigen::Matrix4f input, int M, int N)
{
  std::vector<float> array;
  for (int i = 0; i < M; i++)
  {
    for(int j=0;j<N;j++)
    {
        float tmp;
        tmp = input(i,j);
        array.push_back(tmp);
    }
  }
  return array;
}

// Read a depth image with size H x W and save the depth values (in meters) into a float array (in row-major order)
// The depth image file is assumed to be in 16-bit PNG format, depth in millimeters
void ReadDepth(std::string filename, int H, int W, float * depth, int depth_scale) {
  cv::Mat depth_mat = cv::imread(filename, CV_LOAD_IMAGE_UNCHANGED);
  if (depth_mat.empty()) {
    std::cout << "Error: depth image file not read!" << std::endl;
    cv::waitKey(0);
  }
  for (int r = 0; r < H; ++r)
    for (int c = 0; c < W; ++c) {
      depth[r * W + c] = (float)(depth_mat.at<unsigned short>(r, c)) / depth_scale;
      if (depth[r * W + c] > 6.0f) // Only consider depth < 6m
        depth[r * W + c] = 0;
    }
}

void ReadDepth(cv::Mat depth_mat, int H, int W, float * depth, int depth_scale)
{
  if (depth_mat.empty())
  {
    std::cout << "Error: depth image file not read!" << std::endl;
    cv::waitKey(0);
  }
  for (int r = 0; r < H; ++r)
    for (int c = 0; c < W; ++c)
    {
      depth[r * W + c] = (float)(depth_mat.at<float>(r, c))/depth_scale;
      //std::cout<<"depth:"<<depth_mat.at<float>(r, c)<<std::endl;
      if ((depth[r * W + c] > 6.0f)||(depth[r * W + c]!= depth[r * W + c])) // Only consider depth < 6m and remove "nan"
        depth[r * W + c] = 0;
    }
}

void ReadRGB(cv::Mat rgb_mat, int H, int W, unsigned char * rgb)
{
  if (rgb_mat.empty())
  {
    std::cout << "Error: rgb image file not read!" << std::endl;
    cv::waitKey(0);
  }
  for (int r = 0; r < H; ++r)
    for (int c = 0; c < W; ++c)
    {
      for(int channel = 0; channel < 3; channel++)
      {
        rgb[r * W * 3 + c * 3 + channel] = rgb_mat.ptr<uchar>(r)[c*3+channel];
      }
    }
}

// 4x4 matrix multiplication (matrices are stored as float arrays in row-major order)
void multiply_matrix(const float m1[16], const float m2[16], float mOut[16]) {
  mOut[0]  = m1[0] * m2[0]  + m1[1] * m2[4]  + m1[2] * m2[8]   + m1[3] * m2[12];
  mOut[1]  = m1[0] * m2[1]  + m1[1] * m2[5]  + m1[2] * m2[9]   + m1[3] * m2[13];
  mOut[2]  = m1[0] * m2[2]  + m1[1] * m2[6]  + m1[2] * m2[10]  + m1[3] * m2[14];
  mOut[3]  = m1[0] * m2[3]  + m1[1] * m2[7]  + m1[2] * m2[11]  + m1[3] * m2[15];

  mOut[4]  = m1[4] * m2[0]  + m1[5] * m2[4]  + m1[6] * m2[8]   + m1[7] * m2[12];
  mOut[5]  = m1[4] * m2[1]  + m1[5] * m2[5]  + m1[6] * m2[9]   + m1[7] * m2[13];
  mOut[6]  = m1[4] * m2[2]  + m1[5] * m2[6]  + m1[6] * m2[10]  + m1[7] * m2[14];
  mOut[7]  = m1[4] * m2[3]  + m1[5] * m2[7]  + m1[6] * m2[11]  + m1[7] * m2[15];

  mOut[8]  = m1[8] * m2[0]  + m1[9] * m2[4]  + m1[10] * m2[8]  + m1[11] * m2[12];
  mOut[9]  = m1[8] * m2[1]  + m1[9] * m2[5]  + m1[10] * m2[9]  + m1[11] * m2[13];
  mOut[10] = m1[8] * m2[2]  + m1[9] * m2[6]  + m1[10] * m2[10] + m1[11] * m2[14];
  mOut[11] = m1[8] * m2[3]  + m1[9] * m2[7]  + m1[10] * m2[11] + m1[11] * m2[15];

  mOut[12] = m1[12] * m2[0] + m1[13] * m2[4] + m1[14] * m2[8]  + m1[15] * m2[12];
  mOut[13] = m1[12] * m2[1] + m1[13] * m2[5] + m1[14] * m2[9]  + m1[15] * m2[13];
  mOut[14] = m1[12] * m2[2] + m1[13] * m2[6] + m1[14] * m2[10] + m1[15] * m2[14];
  mOut[15] = m1[12] * m2[3] + m1[13] * m2[7] + m1[14] * m2[11] + m1[15] * m2[15];
}

// 4x4 matrix inversion (matrices are stored as float arrays in row-major order)
bool invert_matrix(const float m[16], float invOut[16]) {
  float inv[16], det;
  int i;
  inv[0] = m[5]  * m[10] * m[15] -
           m[5]  * m[11] * m[14] -
           m[9]  * m[6]  * m[15] +
           m[9]  * m[7]  * m[14] +
           m[13] * m[6]  * m[11] -
           m[13] * m[7]  * m[10];

  inv[4] = -m[4]  * m[10] * m[15] +
           m[4]  * m[11] * m[14] +
           m[8]  * m[6]  * m[15] -
           m[8]  * m[7]  * m[14] -
           m[12] * m[6]  * m[11] +
           m[12] * m[7]  * m[10];

  inv[8] = m[4]  * m[9] * m[15] -
           m[4]  * m[11] * m[13] -
           m[8]  * m[5] * m[15] +
           m[8]  * m[7] * m[13] +
           m[12] * m[5] * m[11] -
           m[12] * m[7] * m[9];

  inv[12] = -m[4]  * m[9] * m[14] +
            m[4]  * m[10] * m[13] +
            m[8]  * m[5] * m[14] -
            m[8]  * m[6] * m[13] -
            m[12] * m[5] * m[10] +
            m[12] * m[6] * m[9];

  inv[1] = -m[1]  * m[10] * m[15] +
           m[1]  * m[11] * m[14] +
           m[9]  * m[2] * m[15] -
           m[9]  * m[3] * m[14] -
           m[13] * m[2] * m[11] +
           m[13] * m[3] * m[10];

  inv[5] = m[0]  * m[10] * m[15] -
           m[0]  * m[11] * m[14] -
           m[8]  * m[2] * m[15] +
           m[8]  * m[3] * m[14] +
           m[12] * m[2] * m[11] -
           m[12] * m[3] * m[10];

  inv[9] = -m[0]  * m[9] * m[15] +
           m[0]  * m[11] * m[13] +
           m[8]  * m[1] * m[15] -
           m[8]  * m[3] * m[13] -
           m[12] * m[1] * m[11] +
           m[12] * m[3] * m[9];

  inv[13] = m[0]  * m[9] * m[14] -
            m[0]  * m[10] * m[13] -
            m[8]  * m[1] * m[14] +
            m[8]  * m[2] * m[13] +
            m[12] * m[1] * m[10] -
            m[12] * m[2] * m[9];

  inv[2] = m[1]  * m[6] * m[15] -
           m[1]  * m[7] * m[14] -
           m[5]  * m[2] * m[15] +
           m[5]  * m[3] * m[14] +
           m[13] * m[2] * m[7] -
           m[13] * m[3] * m[6];

  inv[6] = -m[0]  * m[6] * m[15] +
           m[0]  * m[7] * m[14] +
           m[4]  * m[2] * m[15] -
           m[4]  * m[3] * m[14] -
           m[12] * m[2] * m[7] +
           m[12] * m[3] * m[6];

  inv[10] = m[0]  * m[5] * m[15] -
            m[0]  * m[7] * m[13] -
            m[4]  * m[1] * m[15] +
            m[4]  * m[3] * m[13] +
            m[12] * m[1] * m[7] -
            m[12] * m[3] * m[5];

  inv[14] = -m[0]  * m[5] * m[14] +
            m[0]  * m[6] * m[13] +
            m[4]  * m[1] * m[14] -
            m[4]  * m[2] * m[13] -
            m[12] * m[1] * m[6] +
            m[12] * m[2] * m[5];

  inv[3] = -m[1] * m[6] * m[11] +
           m[1] * m[7] * m[10] +
           m[5] * m[2] * m[11] -
           m[5] * m[3] * m[10] -
           m[9] * m[2] * m[7] +
           m[9] * m[3] * m[6];

  inv[7] = m[0] * m[6] * m[11] -
           m[0] * m[7] * m[10] -
           m[4] * m[2] * m[11] +
           m[4] * m[3] * m[10] +
           m[8] * m[2] * m[7] -
           m[8] * m[3] * m[6];

  inv[11] = -m[0] * m[5] * m[11] +
            m[0] * m[7] * m[9] +
            m[4] * m[1] * m[11] -
            m[4] * m[3] * m[9] -
            m[8] * m[1] * m[7] +
            m[8] * m[3] * m[5];

  inv[15] = m[0] * m[5] * m[10] -
            m[0] * m[6] * m[9] -
            m[4] * m[1] * m[10] +
            m[4] * m[2] * m[9] +
            m[8] * m[1] * m[6] -
            m[8] * m[2] * m[5];

  det = m[0] * inv[0] + m[1] * inv[4] + m[2] * inv[8] + m[3] * inv[12];

  if (det == 0)
    return false;

  det = 1.0 / det;

  for (i = 0; i < 16; i++)
    invOut[i] = inv[i] * det;

  return true;
}

void FatalError(const int lineNumber) {
  std::cerr << "FatalError";
  if (lineNumber != 0) std::cerr << " at LINE " << lineNumber;
  std::cerr << ". Program Terminated." << std::endl;
  hipDeviceReset();
  exit(EXIT_FAILURE);
}

void checkCUDA(const int lineNumber, hipError_t status) {
  if (status != hipSuccess) {
    std::cerr << "CUDA failure at LINE " << lineNumber << ": " << status << std::endl;
    FatalError();
  }
}

void RunKernal(float * cam_K, float * cam2world, float * depth_im, unsigned char * rgb_im,
                int im_height, int im_width, int voxel_grid_dim_x, int voxel_grid_dim_y, int voxel_grid_dim_z,
                float voxel_grid_origin_x, float voxel_grid_origin_y, float voxel_grid_origin_z, float voxel_size, float trunc_margin,
                float * voxel_grid_TSDF, float * voxel_grid_weight,
                unsigned char * voxel_grid_rgb, float * voxel_grid_rgb_weight, float * voxel_grid_rgb_diff)
{
    Integrate <<< voxel_grid_dim_z, voxel_grid_dim_y >>>(cam_K, cam2world, depth_im, rgb_im,
                                                     im_height, im_width, voxel_grid_dim_x, voxel_grid_dim_y, voxel_grid_dim_z,
                                                     voxel_grid_origin_x, voxel_grid_origin_y, voxel_grid_origin_z, voxel_size, trunc_margin,
                                                     voxel_grid_TSDF, voxel_grid_weight, voxel_grid_rgb, voxel_grid_rgb_weight, voxel_grid_rgb_diff);
}


